#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include "Values.h"
#include <GL/glut.h>
#include <thread>
#include <chrono>
extern "C" {
	float *x, *y, *z, *m, *r, *g, *b, *fX, *fY, *fZ;

	//Newton calculations
	__global__
	void newton(float *x,float *y,float *z,float *m,float *r,float *g,float *b,float *fX,float *fY,float *fZ) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;
		for (int i = index; i < N_THREAD; i += stride){
			int a = i / N1;
			int b;
			int c = i % N1;
			if(c < a){
				b = c;
			}
			else{
				b = c+1;
			}
			float dx = x[b] - x[a];
			float dy = y[b] - y[a];
			float dz = z[b] - z[a];
			float squaredDist = 0.0f;
			squaredDist = (dx * dx)+(dy * dy)+(dz*dz);
			if(squaredDist > MIN_SQUARED_DIST){
				float force = G*((m[b]*m[a])/squaredDist);
				//Norme vecteur = distance au carre entre les deux points
				//RENDRE VECTEUR UNITAIRE
				float v0 = dx / squaredDist;
				float v1 = dy / squaredDist;
				float v2 = dz / squaredDist;

				float force_x = v0*force*FAKE_SPEED;
				float force_y = v1*force*FAKE_SPEED;
				float force_z = v2*force*FAKE_SPEED;

				//Ajouter la force dans la "file d'attente"
				fX[a] += force_x;
				fY[a] += force_y;
				fZ[a] += force_z;
			}
		}
	}

	//Called once, at the begining
	__host__ void setupCuda(){

		// Allocate Unified Memory – accessible from CPU or GPU
		hipMallocManaged(&x, NUMBER_OF_PARTICLES*sizeof(float));
		hipMallocManaged(&y, NUMBER_OF_PARTICLES*sizeof(float));
		hipMallocManaged(&z, NUMBER_OF_PARTICLES*sizeof(float));
		hipMallocManaged(&m, NUMBER_OF_PARTICLES*sizeof(float));
		hipMallocManaged(&r, NUMBER_OF_PARTICLES*sizeof(float));
		hipMallocManaged(&g, NUMBER_OF_PARTICLES*sizeof(float));
		hipMallocManaged(&b, NUMBER_OF_PARTICLES*sizeof(float));
		hipMallocManaged(&fX, NUMBER_OF_PARTICLES*sizeof(float));
		hipMallocManaged(&fY, NUMBER_OF_PARTICLES*sizeof(float));
		hipMallocManaged(&fZ, NUMBER_OF_PARTICLES*sizeof(float));

		// initialize x, y, et... arrays on the host
		for (int i = 0; i < NUMBER_OF_PARTICLES; i++) {

		  //RANDOM X, Y, Z  [0, 100]
		  x[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) * RANDOM_RANGE_XYZ;
		  y[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) * RANDOM_RANGE_XYZ;
		  z[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) * RANDOM_RANGE_XYZ;
		  //RANDOM MASS [0, 10]
		  m[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) * MAX_MASS;

		  //RANDOM RGB
		  r[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		  g[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		  b[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);

		  fX[i] = 0.0f;
		  fY[i] = 0.0f;
		  fZ[i] = 0.0f;
	  }
	  std::cout << "Setup on the shared memory done." << '\n';
	  std::cout << "CUDA_N_BLOCKS = " << CUDA_N_BLOCKS<< '\n';
	  std::cout << "NUMBER_OF_PARTICLES = " <<NUMBER_OF_PARTICLES << '\n';
	}

	__host__ int newtonHost() {
		newton<<<CUDA_N_BLOCKS, CUDA_BLOCK_SIZE>>>(x,y,z,m,r,g,b,fX,fY,fZ);
		// Wait for GPU to finish before accessing on host
		hipDeviceSynchronize();
		//Teleport particles at new location
		for (int t = 0; t < NUMBER_OF_PARTICLES; t ++){
			x[t] += fX[t];
			fX[t] = 0.0f;
			y[t] += fY[t];
			fY[t] = 0.0f;
			z[t] += fZ[t];
			fZ[t] = 0.0f;
		}
		//Render the particles as a cube
		for (int i = 0; i < NUMBER_OF_PARTICLES; i++) {
			float x0 = 0.0f+x[i];
		    float x1 = CUBE_SIZE+x[i];
		    float x2 = CUBE_SIZE+x[i];
		    float x3 = 0.0f+x[i];
		    float x4 = 0.0f+x[i];
		    float x5 = 0.0f+x[i];
		    float x6 = CUBE_SIZE+x[i];
		    float x7 = CUBE_SIZE+x[i];

		    float y0 = 0.0f+y[i];
		    float y1 = 0.0f+y[i];
		    float y2 = CUBE_SIZE+y[i];
		    float y3 = CUBE_SIZE+y[i];
		    float y4 = CUBE_SIZE+y[i];
		    float y5 = 0.0f+y[i];
		    float y6 = 0.0f+y[i];
		    float y7 = CUBE_SIZE+y[i];

		    float z0 = 0.0f+z[i];
		    float z1 = 0.0f+z[i];
		    float z2 = 0.0f+z[i];
		    float z3 = 0.0f+z[i];
		    float z4 = CUBE_SIZE+z[i];
		    float z5 = CUBE_SIZE+z[i];
		    float z6 = CUBE_SIZE+z[i];
		    float z7 = CUBE_SIZE+z[i];
		    glColor3f(r[i], g[i], b[i]);
		    glBegin(GL_TRIANGLES);

		    glVertex3f(x0,y0,z0);
		    glVertex3f(x5,y5,z5);
		    glVertex3f(x4,y4,z4);

		    glVertex3f(x0,y0,z0);
		    glVertex3f(x3,y3,z3);
		    glVertex3f(x4,y4,z4);
		    /////////////////////
		    glVertex3f(x3,y3,z3);
		    glVertex3f(x4,y4,z4);
		    glVertex3f(x7,y7,z7);

		    glVertex3f(x3,y3,z3);
		    glVertex3f(x2,y2,z2);
		    glVertex3f(x7,y7,z7);
		    /////////////////////
		    glVertex3f(x5,y5,z5);
		    glVertex3f(x4,y4,z4);
		    glVertex3f(x7,y7,z7);

		    glVertex3f(x5,y5,z5);
		    glVertex3f(x6,y6,z6);
		    glVertex3f(x7,y7,z7);
		    /////////////////////
		    glVertex3f(x1,y1,z1);
		    glVertex3f(x2,y2,z2);
		    glVertex3f(x7,y7,z7);

		    glVertex3f(x1,y1,z1);
		    glVertex3f(x6,y6,z6);
		    glVertex3f(x7,y7,z7);
		    /////////////////////
		    glVertex3f(x0,y0,z0);
		    glVertex3f(x5,y5,z5);
		    glVertex3f(x1,y1,z1);

		    glVertex3f(x6,y6,z6);
		    glVertex3f(x5,y5,z5);
		    glVertex3f(x1,y1,z1);
		    /////////////////////
		    glVertex3f(x0,y0,z0);
		    glVertex3f(x3,y3,z3);
		    glVertex3f(x1,y1,z1);

		    glVertex3f(x3,y3,z3);
		    glVertex3f(x2,y2,z2);
		    glVertex3f(x1,y1,z1);
		    glEnd();
		}

		/*hipFree(x);
		hipFree(y);
		hipFree(z);
		hipFree(m);
		hipFree(r);
		hipFree(g);
		hipFree(b);
		hipFree(fX);
		hipFree(fY);
		hipFree(fZ);*/
		return 0;
	}
}
